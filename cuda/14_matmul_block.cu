
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdlib>
#include <cstdio>
#include <sys/time.h>

#define M 512

__global__ void matmul(float *A, float *B, float *C, int N) {
  int i = blockIdx.x / (N / M);
  int j = threadIdx.x + blockDim.x * (blockIdx.x % (N / M));
  for (int k=0; k<N; k++) {
    C[N*i+j] += A[N*i+k] * B[N*k+j];
  }
}

int main(int argc, char **argv) {
  int N = atoi(argv[1]);
  float * h_A = new float [N*N];
  float * h_B = new float [N*N];
  float * h_C = new float [N*N];
  float *d_A, *d_B, *d_C;
  int size = N * N * sizeof(float);
  hipMalloc((void **) &d_A, size);
  hipMalloc((void **) &d_B, size);
  hipMalloc((void **) &d_C, size);

  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      h_A[N*i+j] = drand48();
      h_B[N*i+j] = drand48();
      h_C[N*i+j] = 0;
    }
  }
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
  struct timeval tic, toc;
  gettimeofday(&tic, NULL);
  matmul<<<N*N/M,M>>>(d_A, d_B, d_C, N);
  hipDeviceSynchronize();
  gettimeofday(&toc, NULL);
  double time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_B, d_B, size, hipMemcpyDeviceToHost);
  hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
  gettimeofday(&tic, NULL);
#pragma omp parallel for
  for (int i=0; i<N; i++) {
    for (int k=0; k<N; k++) {
      for (int j=0; j<N; j++) {
        h_C[N*i+j] -= h_A[N*i+k] * h_B[N*k+j];
      }
    }
  }
  gettimeofday(&toc, NULL);
  time = toc.tv_sec-tic.tv_sec+(toc.tv_usec-tic.tv_usec)*1e-6;
  printf("N=%d: %lf s (%lf GFlops)\n",N,time,2.*N*N*N/time/1e9);
  float err = 0;
  for (int i=0; i<N; i++) {
    for (int j=0; j<N; j++) {
      err += fabs(h_C[N*i+j]);
    }
  }
  printf("error: %f\n",err/N/N);
  delete[] h_A;
  delete[] h_B;
  delete[] h_C;
}
